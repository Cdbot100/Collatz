// Scott Gordon and Steven Kundert
// CMPS 5433 - Colmenares
// Two Loops Project - Collatz Conjecture Verification
// Sequential implementation

#include <hip/hip_runtime.h>
#include <stdio.h>							//standard IO
#include <stdint.h>							//limits file
FILE *f = fopen("VerifyConjecture.txt", "w");//file for writing output
static const int NUM = 1024;

void hailStoneArray(long * a, long size)	//our Sequential code 
{
	long i;
	long iter = 0;							//init values 
	unsigned long long n;					//n can grow very large 
	
	for(i = 2; i < size; i++)				//starting at two, loop for the problem size
	{
		iter = 0;							//set count to zero
		n = i;								//which value are we computing?
		while(n != 1)						//while not converging to 1
		{
			iter++;							//add one more interation
			if((n % 2) == 0)				//if even
			{
				n = n / 2;					// n / 2
			}
			else							//if odd
			{								//
				n = (3 * n + 1) / 2;		//(3n+1)/2. since (3n+1) % 2 == 0, divide by 2
				iter++;						//add one iteration for shortcutted step
			}
		}	
		a[i] = iter;						//store the count in the correct place in the array. 
	}
}

int main()									//our main
{
	const long Asize = (sizeof(long) *NUM);				//size of array (also largest N) 
	long * a_h;							//pointer for array
	long * a_d;
	a_h = (long *)malloc(Asize);//allocate array with zeros, make it correct size
	hailStoneArray(a_h, NUM);			//call our function 
	for (int x = 2; x < NUM; x++)			//loop for output
	{
		fprintf(f,"It takes %d iterations for %d to reach 1 using the Collatz Conjecture\n", a_h[x], x);
	}										//print output
	
	free(a_h); 							//free resources 
	return 0;  								//return 
}
