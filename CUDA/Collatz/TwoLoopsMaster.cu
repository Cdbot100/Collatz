#include "hip/hip_runtime.h"
// Scott Gordon and Steven Kundert
// CMPS 5433 - Colmenares
// Two Loops Project - Collatz Conjecture Verification
// Sequential implementation
#include <stdio.h>								//standard IO
#include <stdint.h>								//limits file
#include<math.h>
#include "timer.h"
FILE *f = fopen("VerifyConjecture.txt", "w");	//file for writing output
static const long NUM = pow(2,31);

__global__ void CUDAhailStoneModified(long * a, long size){
	long i;
	long iter=0;								//init values 
	__shared__ unsigned long long n;			//n can grow very large 

	for (i=threadIdx.x + blockDim.x  * blockIdx.x; i < size; i+= blockDim.x * gridDim.x){
		if (i < size && i >= 2){
			__syncthreads();
			iter = 0;							//set count to zero
			n = i;								//which value are we computing?
			while(n != 1)						//while not converging to 1
			{
				iter++;							//add one more interation
				if((n % 2) == 0)				//if even
				{
					n = n / 2;					// n / 2
				}
				else							//if odd
				{								//
					n = (3 * n + 1) / 2;		//(3n+1)/2. since (3n+1) % 2 == 0, divide by 2
					iter++;						//add one iteration for shortcutted step
				}
			}
			__syncthreads();	
			a[i] = iter;						//store the count in the correct place in the array. 
		}
	}
}

__global__ void CUDAhailStoneArray(long * a, long size){
	long i;
	long iter = 0;								//init values 
	unsigned long long n;						//n can grow very large 

	for (i=threadIdx.x + blockDim.x  * blockIdx.x; i < size; i+= blockDim.x * gridDim.x){
		if (i < size && i >= 2){
			iter = 0;								//set count to zero
				n = i;								//which value are we computing?
				while(n != 1)						//while not converging to 1
				{
					iter++;							//add one more interation
					if((n % 2) == 0)				//if even
					{
						n = n / 2;					// n / 2
					}
					else							//if odd
					{								//
						n = (3 * n + 1) / 2;		//(3n+1)/2. since (3n+1) % 2 == 0, divide by 2
						iter++;						//add one iteration for shortcutted step
					}
				}	
				a[i] = iter;						//store the count in the correct place in the array. 
		}
	}
}

void hailStoneArray(long * a, long size)	//our Sequential code 
{
	long i;
	long iter = 0;							//init values 
	unsigned long long n;					//n can grow very large 
	
	for(i = 2; i < size; i++)				//starting at two, loop for the problem size
	{
		iter = 0;							//set count to zero
		n = i;								//which value are we computing?
		while(n != 1)						//while not converging to 1
		{
			iter++;							//add one more interation
			if((n % 2) == 0)				//if even
			{
				n = n / 2;					// n / 2
			}
			else							//if odd
			{								//
				n = (3 * n + 1) / 2;		//(3n+1)/2. since (3n+1) % 2 == 0, divide by 2
				iter++;						//add one iteration for shortcutted step
			}
		}	
		a[i] = iter;						//store the count in the correct place in the array. 
	}
}

int main()									//our main
{
	double timeCompStart, timeCompEnd, timeComp;
	double timeCommStart, timeCommEnd, timeComm;
	const long Asize = (sizeof(long) *NUM);		//size of array 
	long * a_h, *a_d;							//pointer for array
	a_h = (long *)malloc(Asize);				//allocate array with zeros, make it correct size

	hipMalloc((long**)&a_d,Asize);
	GET_TIME(timeCommStart);
	hipMemcpy(a_d, a_h, Asize, hipMemcpyHostToDevice);
	GET_TIME(timeCommEnd);
	timeComm = timeCommEnd - timeCommStart;

	GET_TIME(timeCompStart);
	CUDAhailStoneArray<<<32, 32>>> (a_d, NUM);
	//CUDAhailStoneModified<<<32,32>>> (a_d, NUM);
	//hailStoneArray(a_h, NUM);	
	hipDeviceSynchronize();
	GET_TIME(timeCompEnd);

	GET_TIME(timeCommStart);
	hipMemcpy(a_h, a_d, Asize, hipMemcpyDeviceToHost);
	GET_TIME(timeCommEnd);

	timeComp = timeCompEnd - timeCompStart;
	timeComm = timeComm + (timeCommEnd - timeCommStart);
	//for (int x = 2; x < NUM; x++)				//loop for output
	//{
		fprintf(f,"It takes %d iterations for %d to reach 1 using the Collatz Conjecture\n", a_h[NUM-1], NUM-1);
	//}											//print output
	
	free(a_h); 									//free resources 
	
	printf ("\telapsed wall clock time: %f s\n", (timeComp+timeComm));

	return 0;  									//return 
}
